#include "hip/hip_runtime.h"
#include <iostream>  
#include "hip/hip_runtime.h"
#include ""

#define	N (512*512)		

__global__ void kernel(float* data)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    data[idx] = idx * idx;
}

int main(int argc, char* argv[])
{
    float   a[N];
    float* dev = NULL;

    hipMalloc((void**)&dev, N * sizeof(float));

    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("GPU : %.10f ms\n\n", gpuTime);

    hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev);


    hipEvent_t start_, stop_;
    float gpuTime_ = 0.0f;
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
    hipEventRecord(start_, 0);

    for (int i = 0; i < N; i++)
    {
        a[i] = i * i;
    }
    hipEventRecord(stop_, 0);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&gpuTime_, start_, stop_);
    printf("CPU : %.10f ms\n\n", gpuTime_);

    return 0;
}
