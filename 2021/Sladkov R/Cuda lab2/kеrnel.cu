
#include "hip/hip_runtime.h"


#include <omp.h>
#include <stdio.h>

hipError_t addWithCuda(unsigned int size);

__global__ void addKernel()
{
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int answ = 1;

    for (int j = 2; j <= i / 2; j++) {
        if (i % j == 0) {
            answ += j;
        }
    }

    if (i == answ && i != 1)
        printf("%d\n", i);
}

int main()
{
    const int arraySize = 100000;
    
    printf("N is %d\n\n", arraySize);

    hipError_t cudaStatus = addWithCuda(arraySize);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
  
    double start;
    double stop;
    start = omp_get_wtime(); 
    
    for (int i = 2; i < arraySize; i++)
    {
        unsigned int answ = 1;

        for (int j = 2; j <= i / 2; j++) {
            if (i % j == 0) {
                answ += j;
            }
        }

        if (i == answ)
            printf("%d\n", i);
    }

    stop = omp_get_wtime();

    printf("Timing CPU Events %.10f", (stop - start) * 1000);

    return 0;
}

hipError_t addWithCuda(unsigned int size)
{
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);

    addKernel<<<(size + 1023) / 1024, 1024>>>();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    hipEventRecord(stop, 0);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("Timing CUDA Events %.10f\n\n", gpuTime);
 
Error:
   
    return cudaStatus;
}
