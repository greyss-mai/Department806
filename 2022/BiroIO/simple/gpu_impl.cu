#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


#define THREAD_NUM 1024


__global__ void cuda_sqrt(float* data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] = sqrt(data[idx]);
}


int GPU_sqrt(float* array, int N) {
           
    int num_bytes = N * sizeof(float);
    int num_blocks = (N < THREAD_NUM) ? 1 : (N / THREAD_NUM);

    dim3 threads(THREAD_NUM);
    dim3 blocks(num_blocks);

    // Memory allocate
    hipEvent_t start, stop;
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    hipEventRecord(start);

    float* in_dev;
    {
        checkCudaErrors(hipMalloc(&in_dev, num_bytes));
        checkCudaErrors(hipMemcpy(in_dev, array, num_bytes, hipMemcpyHostToDevice));
    }

    
    cuda_sqrt<<<blocks, threads>>>(in_dev);
    hipDeviceSynchronize();

    // Memory deallocate
    {
        checkCudaErrors(hipMemcpy(array, in_dev, num_bytes, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(in_dev));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    int microseconds = static_cast<int>(1000.f * milliseconds);

    return microseconds;

}
