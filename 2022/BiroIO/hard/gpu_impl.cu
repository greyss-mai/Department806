#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

// For max thread num:
// 1024 -- 32
// 512  -- 16
// 256  -- 8
#define BLOCK_SIZE 32

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}



__global__ void cuda_mat_add(float* dst, float* mat1, float* mat2, int cols) {
    int idx = (blockIdx.y * blockDim.y + threadIdx.y) * cols + blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = mat1[idx] + mat2[idx];    
}


int GPU_mat_add(float* dst, float* mat1, float* mat2, int N, int M) {

    int sizeof_matrix = N * M * sizeof(float);

    int block_size_x = (BLOCK_SIZE > N) ? 1 : N / BLOCK_SIZE;
    int block_size_y = (BLOCK_SIZE > M) ? 1 : M / BLOCK_SIZE;

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(block_size_x, block_size_y);

    // Memory allocate
    hipEvent_t start, stop;
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    hipEventRecord(start);

    float* out_dev;
    float* in1_dev;
    float* in2_dev;
    {
        checkCudaErrors(hipMalloc(&in1_dev, sizeof_matrix));
        checkCudaErrors(hipMemcpy(in1_dev, mat1, sizeof_matrix, hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc(&in2_dev, sizeof_matrix));
        checkCudaErrors(hipMemcpy(in2_dev, mat2, sizeof_matrix, hipMemcpyHostToDevice));

        checkCudaErrors(hipMalloc(&out_dev, sizeof_matrix));
    }
    
    {
        cuda_mat_add<<<blocks, threads>>>(out_dev, in1_dev, in2_dev, M);
        hipDeviceSynchronize();

        checkCudaErrors(hipGetLastError());
    }

    // Memory deallocate
    {
        checkCudaErrors(hipMemcpy(dst, out_dev, sizeof_matrix, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(out_dev));
        checkCudaErrors(hipFree(in1_dev));
        checkCudaErrors(hipFree(in2_dev));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize(); 
    checkCudaErrors(hipGetLastError());

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    int microseconds = static_cast<int>(1000.f * milliseconds);

    return microseconds;

}
