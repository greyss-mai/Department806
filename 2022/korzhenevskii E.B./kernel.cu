#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <time.h>


#define	N	(1024*1024)

__global__ void kernel(float* data)
{
    int   idx = blockIdx.x * blockDim.x + threadIdx.x;
    float x = 2.0f * 3.1415926f * (float)idx / (float)N;
    data[idx] = sinf(x);
}

int main(int argc, char* argv[])
{
    //CPU

    int start2, time2;
    float* data2 = new float[N];

    start2 = clock();

    for (int idx2 = 0; idx2 < N; idx2++)
    {
        float x2 = 2.0f * 3.1415926f * (float)idx2 / (float)N;
        data2[idx2] = sinf(x2);
    }

    time2 = clock() - start2;
    double time_CPU = time2;

    printf("\nCPU Time: %f milliseconds\n", time_CPU);

    //GPU

    float* a = new float[N];
    float* dev = NULL;

    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMalloc((void**)&dev, N * sizeof(float));

    kernel << <dim3((N / 512), 1), dim3(512, 1) >> > (dev);

    hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("GPU Time: %.2f milliseconds\n", gpuTime);

    hipFree(dev);

    //Print solution

    printf("\nTest values:\n");
    printf("============================");

    int idx = 0;
    printf("\nValue at point (zero):\na[%d] = %.5f\n", idx, a[idx]);

    idx = N / 12;
    printf("\nValue at point (Pi/6):\na[%d] = %.5f\n", idx, a[idx]);

    idx = N / 8;
    printf("\nValue at point (Pi/4):\na[%d] = %.5f\n", idx, a[idx]);

    idx = N / 6;
    printf("\nValue at point (Pi/3):\na[%d] = %.5f\n", idx, a[idx]);

    idx = N / 4;
    printf("\nValue at point (Pi/2):\na[%d] = %.5f\n", idx, a[idx]);

    printf("============================");

    return 0;
}